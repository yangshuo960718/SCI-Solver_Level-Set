#include "hip/hip_runtime.h"
#include <meshFIM3d.h>
#include <tetmesh.h>
#include <Vec.h>
#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <meshFIM_kernels.h>
#include <cutil.h>
#include <cusp/detail/format_utils.h>
#include <cusp/print.h>
#include <thrust/functional.h>
#include <sstream>

extern "C"
{
#include <metis.h>
}

void meshFIM3d::writeFLD()
{
  int nv = m_meshPtr->vertices.size();
  int nt = m_meshPtr->tets.size();
  FILE* fldfile;
  fldfile = fopen("result.fld", "w+");
  fprintf(fldfile, "SCI\nASC\n2\n{@1 {GenericField<TetVolMesh<TetLinearLgn<Point>>,ConstantBasis<float>,vector<float>> 3 {Field 3 {PropertyManager 2 0 }\n}\n{@2 {TetVolMesh<TetLinearLgn<Point>> 4 {Mesh 2 {PropertyManager 2 0 }\n}\n");
  fprintf(fldfile, "{STLVector 2 %d ", nv);
  for(int i = 0; i < nv; i++)
  {
    fprintf(fldfile, "{%.12f %.12f %.12f}", m_meshPtr->vertices[i][0], m_meshPtr->vertices[i][1], m_meshPtr->vertices[i][2]);
  }
  fprintf(fldfile, "}\n{STLIndexVector 1 %d 8 ", nt * 4);
  for(int i = 0; i < nt; i++)
  {
    fprintf(fldfile, "%d %d %d %d ", m_meshPtr->tets[i][0], m_meshPtr->tets[i][1], m_meshPtr->tets[i][2], m_meshPtr->tets[i][3]);
  }
  fprintf(fldfile, "}\n");
  fprintf(fldfile, "{TetLinearLgn<Point>  1 }\n}\n}{ConstantBasis<float>  1 }\n");
  fprintf(fldfile, "{STLVector 2 %d ", nt);
  for(int i = 0; i < nt; i++)
  {
    fprintf(fldfile, " 0");
  }
  fprintf(fldfile, "}\n}\n}");
  fclose(fldfile);
}

void meshFIM3d::writeVTK(std::vector < std::vector <float> > values)
{
  int nv = m_meshPtr->vertices.size();
  int nt = m_meshPtr->tets.size();
  for (size_t j = 0; j < values.size(); j++) {
    FILE* vtkfile;
    std::stringstream ss;
    ss << "result" << j << ".vtk";
    vtkfile = fopen(ss.str().c_str(), "w+");
    fprintf(vtkfile, "# vtk DataFile Version 3.0\nvtk output\nASCII\nDATASET UNSTRUCTURED_GRID\n");
    fprintf(vtkfile, "POINTS %d float\n", nv);
    for (int i = 0; i < nv; i++)
    {
      fprintf(vtkfile, "%.12f %.12f %.12f\n", m_meshPtr->vertices[i][0], m_meshPtr->vertices[i][1], m_meshPtr->vertices[i][2]);
    }
    fprintf(vtkfile, "CELLS %d %d\n", nt, nt * 5);
    for (int i = 0; i < nt; i++)
    {
      fprintf(vtkfile, "4 %d %d %d %d\n", m_meshPtr->tets[i][0], m_meshPtr->tets[i][1], m_meshPtr->tets[i][2], m_meshPtr->tets[i][3]);
    }

    fprintf(vtkfile, "CELL_TYPES %d\n", nt);
    for (int i = 0; i < nt; i++)
    {
      fprintf(vtkfile, "10\n");
    }
    fprintf(vtkfile, "POINT_DATA %d\nSCALARS traveltime float 1\nLOOKUP_TABLE default\n",
        nv, values.size());
    for (int i = 0; i < values[j].size(); i++) {
      fprintf(vtkfile, "%.12f\n ", values[j][i]);
    }
    fclose(vtkfile);
  }
}

void meshFIM3d::updateT_single_stage_d(float timestep, int niter, IdxVector_d& narrowband, int num_narrowband)
{
  int nn = m_meshPtr->vertices.size();
  int nblocks = num_narrowband;
  int nthreads = largest_ele_part;
  thrust::fill(vertT_out.begin(), vertT_out.end(), 0.0);
  int shared_size = sizeof(float)* 4 * largest_ele_part + sizeof(short)*largest_vert_part*m_largest_num_inside_mem;
  cudaSafeCall((kernel_updateT_single_stage3d << <nblocks, nthreads, shared_size >> >(timestep, CAST(narrowband), largest_ele_part, largest_vert_part, full_num_ele,
          CAST(m_ele_after_permute_d), CAST(m_ele_offsets_d), CAST(m_cadv_local_d),
          nn, CAST(m_vert_offsets_d), CAST(m_vert_after_permute_d), CAST(m_vertT_after_permute_d),
          CAST(m_ele_local_coords_d), m_largest_num_inside_mem, CAST(m_mem_locations), CAST(m_mem_location_offsets),
          CAST(vertT_out))));

  nthreads = largest_vert_part;
  cudaSafeCall((CopyOutBack_levelset3d << <nblocks, nthreads >> >(CAST(narrowband),
          CAST(m_vert_offsets_d), CAST(m_vertT_after_permute_d), CAST(vertT_out))));
}

//Single stage update

void meshFIM3d::updateT_single_stage(float timestep, int nside, int niter, std::vector<int>& narrowband)
{
  vec3 sigma(1.0, 0.0, 1.0);
  float epsilon = 1.0;
  int nv = m_meshPtr->vertices.size();
  int nt = m_meshPtr->tets.size();
  std::vector<float> values(4);
  std::vector<float> up(nv, 0.0);
  std::vector<float> down(nv, 0.0);
  std::vector<vec3> node_grad_phi_up(nv, vec3(0.0, 0.0, 0.0));
  std::vector<float> node_grad_phi_down(nv, 0.0);
  std::vector<float> curv_up(nv, 0.0);


  for(int bandidx = 0; bandidx < narrowband.size(); bandidx++)
  {
    int tidx = narrowband[bandidx];
    for(int j = 0; j < 4; j++)
    {
      values[j] = m_meshPtr->vertT[m_meshPtr->tets[tidx][j]];
    }
    //compute ni normals
    std::vector<vec3> nodes(4);
    nodes[0] = (vec3)m_meshPtr->vertices[m_meshPtr->tets[tidx][0]];
    nodes[1] = (vec3)m_meshPtr->vertices[m_meshPtr->tets[tidx][1]];
    nodes[2] = (vec3)m_meshPtr->vertices[m_meshPtr->tets[tidx][2]];
    nodes[3] = (vec3)m_meshPtr->vertices[m_meshPtr->tets[tidx][3]];
    vec3 v31 = nodes[1] - nodes[3];
    vec3 v32 = nodes[2] - nodes[3];
    vec3 v30 = nodes[0] - nodes[3];
    vec3 crossproduct = v31 CROSS v32;
    float dotproduct = crossproduct DOT v30;
    float volume = fabs(dotproduct) / 6.0;

    //compute inverse of 4 by 4 matrix
    float a11 = nodes[0][0], a12 = nodes[0][1], a13 = nodes[0][2], a14 = 1.0;
    float a21 = nodes[1][0], a22 = nodes[1][1], a23 = nodes[1][2], a24 = 1.0;
    float a31 = nodes[2][0], a32 = nodes[2][1], a33 = nodes[2][2], a34 = 1.0;
    float a41 = nodes[3][0], a42 = nodes[3][1], a43 = nodes[3][2], a44 = 1.0;

    float det =
      a11 * a22 * a33 * a44 + a11 * a23 * a34 * a42 + a11 * a24 * a32 * a43
      + a12 * a21 * a34 * a43 + a12 * a23 * a31 * a44 + a12 * a24 * a33 * a41
      + a13 * a21 * a32 * a44 + a13 * a22 * a34 * a41 + a13 * a24 * a31 * a42
      + a14 * a21 * a33 * a42 + a14 * a22 * a31 * a43 + a14 * a23 * a32 * a41
      - a11 * a22 * a34 * a43 - a11 * a23 * a32 * a44 - a11 * a24 * a33 * a42
      - a12 * a21 * a33 * a44 - a12 * a23 * a34 * a41 - a12 * a24 * a31 * a43
      - a13 * a21 * a34 * a42 - a13 * a22 * a31 * a44 - a13 * a24 * a32 * a41
      - a14 * a21 * a32 * a43 - a14 * a22 * a33 * a41 - a14 * a23 * a31 * a42;

    float b11 = a22 * a33 * a44 + a23 * a34 * a42 + a24 * a32 * a43 - a22 * a34 * a43 - a23 * a32 * a44 - a24 * a33 * a42;
    float b12 = a12 * a34 * a43 + a13 * a32 * a44 + a14 * a33 * a42 - a12 * a33 * a44 - a13 * a34 * a42 - a14 * a32 * a43;
    float b13 = a12 * a23 * a44 + a13 * a24 * a42 + a14 * a22 * a43 - a12 * a24 * a43 - a13 * a22 * a44 - a14 * a23 * a42;
    float b14 = a12 * a24 * a33 + a13 * a22 * a34 + a14 * a23 * a32 - a12 * a23 * a34 - a13 * a24 * a32 - a14 * a22 * a33;

    float b21 = a21 * a34 * a43 + a23 * a31 * a44 + a24 * a33 * a41 - a21 * a33 * a44 - a23 * a34 * a41 - a24 * a31 * a43;
    float b22 = a11 * a33 * a44 + a13 * a34 * a41 + a14 * a31 * a43 - a11 * a34 * a43 - a13 * a31 * a44 - a14 * a33 * a41;
    float b23 = a11 * a24 * a43 + a13 * a21 * a44 + a14 * a23 * a41 - a11 * a23 * a44 - a13 * a24 * a41 - a14 * a21 * a43;
    float b24 = a11 * a23 * a34 + a13 * a24 * a31 + a14 * a21 * a33 - a11 * a24 * a33 - a13 * a21 * a34 - a14 * a23 * a31;


    float b31 = a21 * a32 * a44 + a22 * a34 * a41 + a24 * a31 * a42 - a21 * a34 * a42 - a22 * a31 * a44 - a24 * a32 * a41;
    float b32 = a11 * a34 * a42 + a12 * a31 * a44 + a14 * a32 * a41 - a11 * a32 * a44 - a12 * a34 * a41 - a14 * a31 * a42;
    float b33 = a11 * a22 * a44 + a12 * a24 * a41 + a14 * a21 * a42 - a11 * a24 * a42 - a12 * a21 * a44 - a14 * a22 * a41;
    float b34 = a11 * a24 * a32 + a12 * a21 * a34 + a14 * a22 * a31 - a11 * a22 * a34 - a12 * a24 * a31 - a14 * a21 * a32;

    float b41 = a21 * a33 * a42 + a22 * a31 * a43 + a23 * a32 * a41 - a21 * a32 * a43 - a22 * a33 * a41 - a23 * a31 * a42;
    float b42 = a11 * a32 * a43 + a12 * a33 * a41 + a13 * a31 * a42 - a11 * a33 * a42 - a12 * a31 * a43 - a13 * a32 * a41;
    float b43 = a11 * a23 * a42 + a12 * a21 * a43 + a13 * a22 * a41 - a11 * a22 * a43 - a12 * a23 * a41 - a13 * a21 * a42;
    float b44 = a11 * a22 * a33 + a12 * a23 * a31 + a13 * a21 * a32 - a11 * a23 * a32 - a12 * a21 * a33 - a13 * a22 * a31;

    std::vector<vec4> Arows(4);
    Arows[0] = vec4(b11 / det, b12 / det, b13 / det, b14 / det);
    Arows[1] = vec4(b21 / det, b22 / det, b23 / det, b24 / det);
    Arows[2] = vec4(b31 / det, b32 / det, b33 / det, b34 / det);
    Arows[3] = vec4(b41 / det, b42 / det, b43 / det, b44 / det);

    std::vector<vec3> nablaN(4);
    for(int i = 0; i < 4; i++)
    {
      vec4 RHS(0.0, 0.0, 0.0, 0.0);
      RHS[i] = 1.0;
      nablaN[i][0] = Arows[0] DOT RHS;
      nablaN[i][1] = Arows[1] DOT RHS;
      nablaN[i][2] = Arows[2] DOT RHS;
    }

    //compuate grad of Phi
    vec3 nablaPhi(0.0, 0.0, 0.0);
    for(int i = 0; i < 4; i++)
    {
      nablaPhi[0] += nablaN[i][0] * values[i];
      nablaPhi[1] += nablaN[i][1] * values[i];
      nablaPhi[2] += nablaN[i][2] * values[i];
    }
    float abs_nabla_phi = len(nablaPhi);

    //compute K and Kplus and Kminus
    std::vector<float> Kplus(4);
    std::vector<float> Kminus(4);
    std::vector<float> K(4);
    float Hintegral = 0.0;
    float beta = 0;
    for(int i = 0; i < 4; i++)
    {
      K[i] = volume * (sigma DOT nablaN[i]); // for H(\nabla u) = sigma DOT \nabla u
      Hintegral += K[i] * values[i];
      Kplus[i] = fmax(K[i], (float)0.0);
      Kminus[i] = fmin(K[i], (float)0.0);
      beta += Kminus[i];
    }
    beta = 1.0 / beta;

    if(fabs(Hintegral) > 1e-16)
    {
      std::vector<float> delta(4);
      for(int i = 0; i < 4; i++)
      {
        delta[i] = Kplus[i] * beta * (Kminus[0] * (values[i] - values[0]) + Kminus[1] * 
          (values[i] - values[1]) + Kminus[2] * (values[i] - values[2]) + Kminus[3] * (values[i] - values[3]));
      }

      std::vector<float> alpha(4);
      for(int i = 0; i < 4; i++)
      {
        alpha[i] = delta[i] / Hintegral;
      }

      float theta = 0;
      for(int i = 0; i < 4; i++)
      {
        theta += fmax((float)0.0, alpha[i]);
      }

      std::vector<float> alphatuda(4);
      for(int i = 0; i < 4; i++)
      {
        alphatuda[i] = fmax(alpha[i], (float)0.0) / theta;
      }

      for(int i = 0; i < 4; i++)
      {
        up[m_meshPtr->tets[tidx][i]] += alphatuda[i] * Hintegral;
        down[m_meshPtr->tets[tidx][i]] += alphatuda[i] * volume;
        node_grad_phi_up[m_meshPtr->tets[tidx][i]] += volume* nablaPhi;
        node_grad_phi_down[m_meshPtr->tets[tidx][i]] += volume;
        curv_up[m_meshPtr->tets[tidx][i]] += volume * ((nablaN[i] DOT nablaN[i]) / abs_nabla_phi * values[i] +
            (nablaN[i] DOT nablaN[(i + 1) % 4]) / abs_nabla_phi * values[(i + 1) % 4] +
            (nablaN[i] DOT nablaN[(i + 2) % 4]) / abs_nabla_phi * values[(i + 2) % 4] +
            (nablaN[i] DOT nablaN[(i + 3) % 4]) / abs_nabla_phi * values[(i + 3) % 4]);
      }
    }
  }

  for(int vidx = 0; vidx < nv; vidx++)
  {
    float eikonal = up[vidx] / down[vidx];
    float curvature = curv_up[vidx] / node_grad_phi_down[vidx];
    float node_eikonal = len(node_grad_phi_up[vidx]) / node_grad_phi_down[vidx];
    if(fabs(down[vidx]) > 1e-16)
    {
      m_meshPtr->vertT[vidx] -= epsilon * node_eikonal * curvature * timestep;
    }
  }
}

void meshFIM3d::GraphPartition_Square(int squareLength, int squareWidth, int squareHeight,
    int blockLength, int blockWidth, int blockHeight, bool verbose)
{
  int nn = m_meshPtr->vertices.size();
  int numBlockLength = ceil((float)squareLength / blockLength);
  int numBlockWidth = ceil((float)squareWidth / blockWidth);
  int numBlockHeight = ceil((float)squareHeight / blockHeight);
  int numBlock = numBlockLength * numBlockWidth*numBlockHeight;
  npart_h = IdxVector_h(nn);
  nparts = numBlock;

  int edgeCount = 0;
  for(int vIt = 0; vIt < nn; vIt++)
  {
    edgeCount += m_meshPtr->neighbors[vIt].size();
  }

  m_largest_num_inside_mem = 0;
  for(int i = 0; i < nn; i++)
  {
    if(m_meshPtr->adjacenttets[i].size() > m_largest_num_inside_mem)
      m_largest_num_inside_mem = m_meshPtr->adjacenttets[i].size();
  }
  if (verbose)
    printf("m_largest_num_inside_mem = %d\n", m_largest_num_inside_mem);

  //Allocating storage for array values of adjacency
  int* xadj = new int[nn + 1];
  int* adjncy = new int[edgeCount];

  // filling the arrays:
  xadj[0] = 0;
  int idx = 0;
  IdxVector_h neighbor_sizes(nn);
  // Populating the arrays:
  for(int i = 1; i < nn + 1; i++)
  {
    neighbor_sizes[i - 1] = m_meshPtr->neighbors[i - 1].size();
    xadj[i] = xadj[i - 1] + m_meshPtr->neighbors[i - 1].size();
    for(int j = 0; j < m_meshPtr->neighbors[i - 1].size(); j++)
    {
      adjncy[idx++] = m_meshPtr->neighbors[i - 1][j];
    }
  }

  m_neighbor_sizes_d = neighbor_sizes;

  for(int k = 0; k < squareHeight; k++)
    for(int i = 0; i < squareWidth; i++)
      for(int j = 0; j < squareLength; j++)
      {
        int index = k * squareLength * squareWidth + i * squareLength + j;
        int k2 = k;
        int i2 = i;
        int j2 = j;
        npart_h[index] = (k2 / blockHeight) * numBlockLength *
          numBlockWidth + (i2 / blockWidth) * numBlockLength + (j2 / blockLength);
      }

  m_xadj_d = IdxVector_d(&xadj[0], &xadj[nn + 1]);
  m_adjncy_d = IdxVector_d(&adjncy[0], &adjncy[edgeCount]);

  IdxVector_h part_sizes(nparts, 0);
  if (verbose) {
    std::cout << npart_h.size() << std::endl;
    std::cout << part_sizes.size() << std::endl;
    std::cout << nn << std::endl;
  }
  for(int i = 0; i < nn; i++)
  {
    part_sizes[npart_h[i]]++;
  }
  int min_part_size = thrust::reduce(part_sizes.begin(), part_sizes.end(),
      100000000, thrust::minimum<int>());
  largest_vert_part = thrust::reduce(part_sizes.begin(), part_sizes.end(),
      -1, thrust::maximum<int>());
  if(verbose)
    printf("Largest vertex partition size is: %d\n", largest_vert_part);
  if(min_part_size == 0)
    if(verbose)
      printf("Min partition size is 0!!\n");
  delete[] xadj;
  delete[] adjncy;
}

void meshFIM3d::Partition_METIS(int metissize, bool verbose)
{
  int options[10], pnumflag = 0, wgtflag = 0;
  options[0] = 0;
  int edgecut;
  int nn = m_meshPtr->vertices.size();
  npart_h = IdxVector_h(nn);
  nparts = ceil((float)nn / (float)metissize);

  // Counting up edges for adjacency:
  int edgeCount = 0;
  for(int vIt = 0; vIt < nn; vIt++)
  {
    edgeCount += m_meshPtr->neighbors[vIt].size();// m_meshPtr->neighbors[vIt].size()：一个节点周围节点数
  }

  m_largest_num_inside_mem = 0;
  for(int i = 0; i < nn; i++)
  {
    if(m_meshPtr->adjacenttets[i].size() > m_largest_num_inside_mem)
      m_largest_num_inside_mem = m_meshPtr->adjacenttets[i].size();
  }//找一个节点周围最多有多少个单元
  if (verbose)
    printf("m_largest_num_inside_mem = %d\n", m_largest_num_inside_mem);


  //Allocating storage for array values of adjacency
  int* xadj = new int[nn + 1];
  int* adjncy = new int[edgeCount];

  // filling the arrays:
  xadj[0] = 0;
  int idx = 0;
  IdxVector_h neighbor_sizes(nn);
  // Populating the arrays:
  for(int i = 1; i < nn + 1; i++)
  {
    neighbor_sizes[i - 1] = m_meshPtr->neighbors[i - 1].size();
    xadj[i] = xadj[i - 1] + m_meshPtr->neighbors[i - 1].size();
    for(int j = 0; j < m_meshPtr->neighbors[i - 1].size(); j++)
    {
      adjncy[idx++] = m_meshPtr->neighbors[i - 1][j];
    }
  }

  m_neighbor_sizes_d = neighbor_sizes;

  METIS_PartGraphKway(&nn, xadj, adjncy, NULL, NULL, &wgtflag, &pnumflag,
      &nparts, options, &edgecut, thrust::raw_pointer_cast(&npart_h[0]));

  m_xadj_d = IdxVector_d(&xadj[0], &xadj[nn + 1]);
  m_adjncy_d = IdxVector_d(&adjncy[0], &adjncy[edgeCount]);

  IdxVector_h part_sizes(nparts, 0);
  for(int i = 0; i < nn; i++)
  {
    part_sizes[npart_h[i]]++;
  }
  int min_part_size = thrust::reduce(part_sizes.begin(), part_sizes.end(), 100000000, thrust::minimum<int>());
  largest_vert_part = thrust::reduce(part_sizes.begin(), part_sizes.end(), -1, thrust::maximum<int>());
  if (verbose)
    printf("Largest vertex partition size is: %d\n", largest_vert_part);
  if(min_part_size == 0)
    if (verbose)
      printf("Min partition size is 0!!\n");
  delete [] xadj;
  delete [] adjncy;
}

void meshFIM3d::InitPatches(bool verbose)
{
  int ne = m_meshPtr->tets.size();
  int nn = m_meshPtr->vertices.size();
  ele_d = IdxVector_d(4 * ne);
  ele_h = IdxVector_h(4 * ne);
  vert_d = Vector_d(3 * nn);
  m_vert_after_permute_d = Vector_d(3 * nn);
  Vector_h vert_h(3 * nn);
  for(int eidx = 0; eidx < ne; eidx++)
  {
    for(int i = 0; i < 4; i++)
      ele_h[i * ne + eidx] = m_meshPtr->tets[eidx][i]; //interleaved storage
  }
  for(int vidx = 0; vidx < nn; vidx++)
  {
    for(int i = 0; i < 3; i++)
      vert_h[i * nn + vidx] = m_meshPtr->vertices[vidx][i]; //interleaved storage
  }
  ele_d = ele_h;
  vert_d = vert_h;
  m_npart_d = IdxVector_d(npart_h.begin(), npart_h.end());
  m_part_label_d = IdxVector_d(m_npart_d.begin(), m_npart_d.end());
  int nthreads = 256;
  int nblocks = min((int)ceil((float)ne / nthreads), 65535);
  cudaSafeCall((kernel_compute_ele_npart3d << <nblocks, nthreads >> >(ne,
          thrust::raw_pointer_cast(&m_npart_d[0]), thrust::raw_pointer_cast(&ele_d[0]),
          thrust::raw_pointer_cast(&ele_label_d[0]))));


  full_num_ele = thrust::reduce(ele_label_d.begin(), ele_label_d.end());
  if(verbose)
    printf("full_num_ele = %d\n", full_num_ele);
  ele_offsets_d[0] = 0;
  thrust::inclusive_scan(ele_label_d.begin(), ele_label_d.end(), ele_offsets_d.begin() + 1);
  ele_full_label = IdxVector_d(full_num_ele);
  ele_permute = IdxVector_d(full_num_ele);

  cudaSafeCall((kernel_fill_ele_label3d << <nblocks, nthreads >> >(ne, thrust::raw_pointer_cast(&ele_permute[0]),
          thrust::raw_pointer_cast(&ele_offsets_d[0]),
          thrust::raw_pointer_cast(&m_npart_d[0]), thrust::raw_pointer_cast(&ele_d[0]),
          thrust::raw_pointer_cast(&ele_full_label[0]))));

  clock_t starttime, endtime;
  double duration;
  starttime = clock();
  thrust::sort_by_key(ele_full_label.begin(), ele_full_label.end(), ele_permute.begin());
  hipDeviceSynchronize();
  endtime = clock();
  duration = (double)(endtime - starttime) / (double)CLOCKS_PER_SEC;
  if(verbose)
    printf("Sorting time : %.10lf s\n", duration);
  m_ele_offsets_d = IdxVector_d(nparts + 1);
  ones = IdxVector_d(full_num_ele, 1);
  tmp = IdxVector_d(full_num_ele);
  reduce_output = IdxVector_d(full_num_ele);
  thrust::reduce_by_key(ele_full_label.begin(),
      ele_full_label.end(), ones.begin(), tmp.begin(), reduce_output.begin());
  largest_ele_part = thrust::reduce(reduce_output.begin(),
      reduce_output.begin() + nparts, -1, thrust::maximum<int>());
  if(verbose)
    printf("Largest element partition size is: %d\n", largest_ele_part);
  if(largest_ele_part > 1024)
  {
    printf("Error: largest_ele_part > 1024 !!\n");
    exit(0);
  }
  m_ele_offsets_d[0] = 0;
  thrust::inclusive_scan(reduce_output.begin(), reduce_output.begin() + nparts,
      m_ele_offsets_d.begin() + 1);
}

void meshFIM3d::InitPatches2()
{
  int ne = m_meshPtr->tets.size();
  int nn = m_meshPtr->vertices.size();
  IdxVector_d vert_permute(nn, 0);
  IdxVector_d vert_ipermute(nn, 0);
  int nthreads = 256;
  int nblocks = min((int)ceil((float)nn / nthreads), 65535);
  cudaSafeCall((kernel_fill_sequence3d << <nblocks, nthreads >> >(nn, CAST(vert_permute))));
  thrust::sort_by_key(m_part_label_d.begin(), m_part_label_d.end(), vert_permute.begin());
  nblocks = min((int)ceil((float)nn / nthreads), 65535);
  cudaSafeCall((kernel_compute_vert_ipermute3d << <nblocks, nthreads >> >(nn,
          thrust::raw_pointer_cast(&vert_permute[0]), thrust::raw_pointer_cast(&vert_ipermute[0]))));

  m_vert_permute_d = IdxVector_d(vert_permute);
  m_vert_offsets_d = IdxVector_d(nparts + 1);
  cusp::detail::indices_to_offsets(m_part_label_d, m_vert_offsets_d);

  //permute the vert and ele values
  m_ele_after_permute_d = IdxVector_d(4 * full_num_ele);
  m_vertT_after_permute_d = Vector_d(nn);
  nblocks = min((int)ceil((float)full_num_ele / nthreads), 65535);
  cudaSafeCall((kernel_ele_and_vert3d << <nblocks, nthreads >> >(full_num_ele, ne,
          thrust::raw_pointer_cast(&ele_d[0]), thrust::raw_pointer_cast(&m_ele_after_permute_d[0]),
          thrust::raw_pointer_cast(&ele_permute[0]),
          nn, thrust::raw_pointer_cast(&vert_d[0]), thrust::raw_pointer_cast(&m_vert_after_permute_d[0]),
          thrust::raw_pointer_cast(&m_vertT_d[0]), thrust::raw_pointer_cast(&m_vertT_after_permute_d[0]),
          CAST(vert_permute),
          thrust::raw_pointer_cast(&vert_ipermute[0]))));

  //compute the local coords for each element
  m_ele_local_coords_d = Vector_d(6 * full_num_ele);
  m_cadv_local_d = Vector_d(3 * full_num_ele);
  nthreads = 256;
  nblocks = min((int)ceil((float)full_num_ele / nthreads), 65535);
  cudaSafeCall((kernel_compute_local_coords3d << <nblocks, nthreads >> >(full_num_ele, nn,
          thrust::raw_pointer_cast(&m_ele_after_permute_d[0]), thrust::raw_pointer_cast(&m_ele_offsets_d[0]),
          thrust::raw_pointer_cast(&m_vert_after_permute_d[0]),
          thrust::raw_pointer_cast(&m_ele_local_coords_d[0]),
          CAST(m_cadv_global_d), CAST(m_cadv_local_d))));
  //Generate redution list

  m_mem_locations = IdxVector_d(4 * full_num_ele);
  IdxVector_d tmp2 = m_ele_after_permute_d;
  thrust::sequence(m_mem_locations.begin(), m_mem_locations.end(), 0);
  thrust::sort_by_key(tmp2.begin(), tmp2.end(), m_mem_locations.begin());
  m_mem_location_offsets = IdxVector_d(nn + 1);
  cusp::detail::indices_to_offsets(tmp2, m_mem_location_offsets);

}

void meshFIM3d::GenerateBlockNeighbors()
{

  //Generate block neighbors
  // Declaring temporary vectors:
  adjacencyBlockLabel = IdxVector_d(m_adjncy_d.size(), 0);
  blockMappedAdjacency = IdxVector_d(m_adjncy_d.size(), 0);

  mapAdjacencyToBlock(m_xadj_d, m_adjncy_d, adjacencyBlockLabel, blockMappedAdjacency, m_npart_d);
  // Zip up the block label and block mapped vectors and sort:
  thrust::sort(thrust::make_zip_iterator(thrust::make_tuple(adjacencyBlockLabel.begin(), blockMappedAdjacency.begin())),
      thrust::make_zip_iterator(thrust::make_tuple(adjacencyBlockLabel.end(), blockMappedAdjacency.end())));

  // Remove Duplicates and resize:
  int newSize = thrust::unique(thrust::make_zip_iterator(thrust::make_tuple(adjacencyBlockLabel.begin(), blockMappedAdjacency.begin())),
      thrust::make_zip_iterator(thrust::make_tuple(adjacencyBlockLabel.end(), blockMappedAdjacency.end()))) -
    thrust::make_zip_iterator(thrust::make_tuple(adjacencyBlockLabel.begin(), blockMappedAdjacency.begin()));


  adjacencyBlockLabel.resize(newSize);
  blockMappedAdjacency.resize(newSize);
  getPartIndicesNegStart(adjacencyBlockLabel, m_block_xadj_d);
  m_block_adjncy_d.resize(blockMappedAdjacency.size() - 1);
  thrust::copy(blockMappedAdjacency.begin() + 1, blockMappedAdjacency.end(), m_block_adjncy_d.begin());
}

std::vector <std::vector <float> > meshFIM3d::GenerateData(
    char* filename, int nsteps, float timestep, int inside_niter,
    int nside, int block_size, float bandwidth, int part_type,
    int metis_size, bool verbose)
{
  if (verbose)
    printf("Starting meshFIM::GenerateData\n");
  int nv = m_meshPtr->vertices.size();
  int nt = m_meshPtr->tets.size();

  int squareLength = nside;
  int squareWidth = nside;
  int squareDepth = nside;
  int squareBlockLength = block_size;
  int squareBlockWidth = block_size;
  int squareBlockDepth = block_size;
  //  float starttime, endtime, duration;
  clock_t starttime, endtime, starttime1, endtime1;
  float duration, duration1 = 0.0, duration2 = 0.0;

  if(part_type == 1)
    GraphPartition_Square(squareLength, squareWidth, squareDepth,
        squareBlockLength, squareBlockWidth, squareBlockDepth, verbose);
  else //partition with METIS
  {
    Partition_METIS(metis_size, verbose);
  }
  //Initialize the values
  Vector_h h_vertT(nv);
  for(int i = 0; i < nv; i++)
  {
    h_vertT[i] = m_meshPtr->vertT[i];
  }
  m_vertT_d = h_vertT;
  starttime = clock();
  //Init patches
  InitPatches(verbose);
  Vector_h cadv_h(3 * full_num_ele,0);
  IdxVector_h ele_permute_h = IdxVector_h(ele_permute);
  for (int i = 0; i < full_num_ele; i++) {
    size_t tetIdx = static_cast<size_t>(ele_permute_h[i]);
    cadv_h[0 * full_num_ele + i] = m_meshPtr->normals[tetIdx][0];
    cadv_h[1 * full_num_ele + i] = m_meshPtr->normals[tetIdx][1];
    cadv_h[2 * full_num_ele + i] = m_meshPtr->normals[tetIdx][2];
  }

  m_cadv_global_d = Vector_d(cadv_h);
  InitPatches2(); 
  GenerateBlockNeighbors();
  if (verbose)
    printf("After  preprocessing\n");
  endtime = clock();
  duration = (float)(endtime - starttime) / CLOCKS_PER_SEC;
  if (verbose)
    printf("pre processing time : %.10lf s\n", duration);

  //Inite redistance
  m_redist = new redistance3d(m_meshPtr, nparts, m_block_xadj_d, m_block_adjncy_d);

  //////////////////////////update values///////////////////////////////////////////
  IdxVector_d narrowband_d(nparts);
  int num_narrowband = 0;

  std::vector <std::vector <float> >  ans;
  ans.push_back(m_meshPtr->vertT);

  starttime = clock();
  for(int stepcount = 0; stepcount < nsteps; stepcount++)
  {
    m_redist->FindSeedPoint(narrowband_d, num_narrowband, m_meshPtr, m_vertT_after_permute_d, nparts, largest_vert_part, largest_ele_part, m_largest_num_inside_mem, full_num_ele,
        m_vert_after_permute_d, m_vert_offsets_d, m_ele_after_permute_d, m_ele_offsets_d, m_ele_local_coords_d, m_mem_location_offsets, m_mem_locations,
        m_part_label_d, m_block_xadj_d, m_block_adjncy_d);

    m_redist->ReInitTsign(m_meshPtr, m_vertT_after_permute_d, nparts, largest_vert_part, largest_ele_part, m_largest_num_inside_mem, full_num_ele,
        m_vert_after_permute_d, m_vert_offsets_d, m_ele_after_permute_d, m_ele_offsets_d, m_ele_local_coords_d, m_mem_location_offsets, m_mem_locations,
        m_part_label_d, m_block_xadj_d, m_block_adjncy_d);
    starttime1 = clock();
    m_redist->GenerateData(narrowband_d, num_narrowband, bandwidth, stepcount, m_meshPtr, m_vertT_after_permute_d, nparts, largest_vert_part, largest_ele_part, m_largest_num_inside_mem, full_num_ele,
        m_vert_after_permute_d, m_vert_offsets_d, m_ele_after_permute_d, m_ele_offsets_d, m_ele_local_coords_d, m_mem_location_offsets, m_mem_locations,
        m_part_label_d, m_block_xadj_d, m_block_adjncy_d, verbose);
    hipDeviceSynchronize();
    endtime1 = clock();
    duration1 += endtime1 - starttime1;
    if (num_narrowband == 0) {
      std::cout << "NOTE: Ending at timestep " << stepcount <<
        " due to zero narrow band." << std::endl;
      break;
    }
    starttime1 = clock();
    for(int niter = 0; niter < inside_niter; niter++)
      updateT_single_stage_d(timestep, stepcount, narrowband_d, num_narrowband);

    hipDeviceSynchronize();
    endtime1 = clock();
    duration2 += endtime1 - starttime1;
    ///////////////////done updating/////////////////////////////////////////////////
    int nthreads = 256;
    int nblocks = min((int)ceil((float)nv / nthreads), 655535);
    cudaSafeCall((kernel_compute_vertT_before_permute3d << <nblocks, nthreads >> >(nv, 
      CAST(m_vert_permute_d), CAST(m_vertT_after_permute_d), CAST(tmp_vertT_before_permute_d))));
    Vector_h vertT_before_permute_h = tmp_vertT_before_permute_d;
    for(int i = 0; i < nv; i++)
    {
      m_meshPtr->vertT[i] = vertT_before_permute_h[i];
    }
    ans.push_back(m_meshPtr->vertT);
  }

  hipDeviceSynchronize();
  endtime = clock();
  if (verbose)
    printf("redistance time : %.10lf s\n", (float)duration1 / CLOCKS_PER_SEC);
  if (verbose)
    printf("levelset update time : %.10lf s\n", (float)duration2 / CLOCKS_PER_SEC);
  duration = (double)(endtime - starttime) / (double)CLOCKS_PER_SEC;
  if (verbose)
    printf("Processing time : %.10lf s\n", duration);
  return ans;
}

void meshFIM3d::getPartIndicesNegStart(IdxVector_d& sortedPartition, IdxVector_d& partIndices)
{
  // Sizing the array:
  int maxPart = sortedPartition[sortedPartition.size() - 1];
  partIndices.resize(maxPart + 2, 0);

  // Figuring out block sizes for kernel call:
  int size = sortedPartition.size();
  int blockSize = 256;
  int nBlocks = size / blockSize + (size % blockSize == 0 ? 0 : 1);

  // Getting pointers
  int *sortedPartition_d = thrust::raw_pointer_cast(&sortedPartition[0]);
  int *partIndices_d = thrust::raw_pointer_cast(&partIndices[0]);

  // Calling kernel to find indices for each part:
  findPartIndicesNegStartKernel3d << < nBlocks, blockSize >> > (size, sortedPartition_d, partIndices_d);
  partIndices[partIndices.size() - 1] = size - 1;
}

void meshFIM3d::mapAdjacencyToBlock(IdxVector_d &adjIndexes, IdxVector_d &adjacency, 
  IdxVector_d &adjacencyBlockLabel, IdxVector_d &blockMappedAdjacency, IdxVector_d &fineAggregate)
{
  int size = adjIndexes.size() - 1;
  // Get pointers:adjacencyIn
  int *adjIndexes_d = thrust::raw_pointer_cast(&adjIndexes[0]);
  int *adjacency_d = thrust::raw_pointer_cast(&adjacency[0]);
  int *adjacencyBlockLabel_d = thrust::raw_pointer_cast(&adjacencyBlockLabel[0]);
  int *blockMappedAdjacency_d = thrust::raw_pointer_cast(&blockMappedAdjacency[0]);
  int *fineAggregate_d = thrust::raw_pointer_cast(&fineAggregate[0]);

  // Figuring out block sizes for kernel call:
  int blockSize = 256;
  int nBlocks = size / blockSize + (size % blockSize == 0 ? 0 : 1);

  // Calling kernel:
  mapAdjacencyToBlockKernel3d << < nBlocks, blockSize >> > (size, adjIndexes_d, adjacency_d, 
    adjacencyBlockLabel_d, blockMappedAdjacency_d, fineAggregate_d);
}

